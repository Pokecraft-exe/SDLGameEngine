#include "hip/hip_runtime.h"
#include "game.h"
#define CUDA_WARN(XXX) \
    do { if (XXX != hipSuccess) cerr << "CUDA Error: " << \
        hipGetErrorString(XXX) << ", at line " << __LINE__ \
        << endl; hipDeviceSynchronize(); } while (0)

typedef vector<vec3d> Vertvec;

__global__ void calculatecuda(vec3d* vert, vec3d* outvert, vec3d* temp, mat4x4* World, mat4x4* matrix, mat4x4* projection, int* w, int* h) {
	int i = threadIdx.x;

	printf("vert[%d] : x = %d y= %d z= %d\n", i, vert[i].x, vert[i].y, vert[i].z);

	temp[0].x = vert[i].x * World->m[0][0] + vert[i].y * World->m[1][0] + vert[i].z * World->m[2][0] + vert[i].w * World->m[3][0];
	temp[0].y = vert[i].x * World->m[0][1] + vert[i].y * World->m[1][1] + vert[i].z * World->m[2][1] + vert[i].w * World->m[3][1];
	temp[0].z = vert[i].x * World->m[0][2] + vert[i].y * World->m[1][2] + vert[i].z * World->m[2][2] + vert[i].w * World->m[3][2];
	temp[0].w = vert[i].x * World->m[0][3] + vert[i].y * World->m[1][3] + vert[i].z * World->m[2][3] + vert[i].w * World->m[3][3];

	temp[1].x = temp[0].x * matrix->m[0][0] + temp[0].y * matrix->m[1][0] + temp[0].z * matrix->m[2][0] + temp[0].w * matrix->m[3][0];
	temp[1].y = temp[0].x * matrix->m[0][1] + temp[0].y * matrix->m[1][1] + temp[0].z * matrix->m[2][1] + temp[0].w * matrix->m[3][1];
	temp[1].z = temp[0].x * matrix->m[0][2] + temp[0].y * matrix->m[1][2] + temp[0].z * matrix->m[2][2] + temp[0].w * matrix->m[3][2];
	temp[1].w = temp[0].x * matrix->m[0][3] + temp[0].y * matrix->m[1][3] + temp[0].z * matrix->m[2][3] + temp[0].w * matrix->m[3][3];

	temp[0] = temp[1];

	temp[1].x = temp[0].x * projection->m[0][0] + temp[0].y * projection->m[1][0] + temp[0].z * projection->m[2][0] + temp[0].w * projection->m[3][0];
	temp[1].y = temp[0].x * projection->m[0][1] + temp[0].y * projection->m[1][1] + temp[0].z * projection->m[2][1] + temp[0].w * projection->m[3][1];
	temp[1].z = temp[0].x * projection->m[0][2] + temp[0].y * projection->m[1][2] + temp[0].z * projection->m[2][2] + temp[0].w * projection->m[3][2];
	temp[1].w = temp[0].x * projection->m[0][3] + temp[0].y * projection->m[1][3] + temp[0].z * projection->m[2][3] + temp[0].w * projection->m[3][3];

	outvert[i].x = temp[1].x / temp[1].w;
	outvert[i].y = temp[1].y / temp[1].w;
	outvert[i].z = temp[1].z / temp[1].w;
	outvert[i].x += 1.0f;
	outvert[i].y += 1.0f;
	outvert[i].x *= 0.5f * (float)*w;
	outvert[i].y *= 0.5f * (float)*h;
}
void calculatePolygonsCuda(__parameters _param, void* _this) {
	Vertvec* _verts = _param.verts;
	Vertvec _outVerts;
	Vertvec _Temp;
	mesh* _mesh = _param._mesh;
	camera* _camera = _param._camera;
	light* _light = _param._light;

	mat4x4 World, Rx, Ry, Rz, t;

	_param.Rx = &Rx;
	_param.Ry = &Ry;
	_param.Rz = &Rz;
	_param.t = &t;
	_param.World = &World;

	((SDLGameEngine*)_this)->_meshApplyRotations(_param);
	((SDLGameEngine*)_this)->_meshApplyTransations(_param);

	World = ((SDLGameEngine*)_this)->_matrixMakeIdentity();
	World = Matrix_MultiplyMatrix(Ry, Rz);
	World = Matrix_MultiplyMatrix(World, Rx);
	World = Matrix_MultiplyMatrix(World, t);

	mat4x4 _cameraRotationX;
	mat4x4 _cameraRotationY;
	mat4x4 _cameraRotationZ;
	mat4x4 _cameraRotation;

	_param.Rx = &_cameraRotationX;
	_param.Ry = &_cameraRotationY;
	_param.Rz = &_cameraRotationZ;

	((SDLGameEngine*)_this)->_cameraApplyRotations(_param);

	_cameraRotation = Matrix_MultiplyMatrix(_cameraRotationZ, _cameraRotationY);
	_cameraRotation = Matrix_MultiplyMatrix(_cameraRotation, _cameraRotationZ);

	_camera->_up = { 0, 1, 0 };
	_camera->_target = { 0, 0, 1 };
	_camera->_pointAt = MultiplyMatrixVector(_cameraRotation, _camera->_target);

	_camera->_target = Vector_Add(_camera->position, _camera->_pointAt);

	mat4x4 __matrix = Matrix_PointAt(_camera->position, _camera->_target, _camera->_up);
	_camera->_matrix = __matrix; Matrix_QuickInverse(__matrix);

	_outVerts.resize(_verts->size());
	_Temp.resize(2);

	mat4x4* gpuWorld;
	mat4x4* gpuMatrix;
	mat4x4* gpuProjection;
	vec3d* gpuVerts;
	vec3d* gpuOutVerts;
	vec3d* gpuTemp;
	int* gpuW;
	int* gpuH;

	size_t size = _mesh->vertices.size();

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	hipMalloc((void**)&gpuWorld, sizeof(mat4x4));
	hipMalloc((void**)&gpuMatrix, sizeof(mat4x4));
	hipMalloc((void**)&gpuProjection, sizeof(mat4x4));
	hipMalloc((void**)&gpuVerts, size * sizeof(vec3d));
	hipMalloc((void**)&gpuOutVerts, size * sizeof(vec3d));
	hipMalloc((void**)&gpuTemp, 2 * sizeof(vec3d));
	hipMalloc((void**)&gpuW, sizeof(int));
	hipMalloc((void**)&gpuH, sizeof(int));
	hipMemcpy(gpuWorld, &World, sizeof(mat4x4), hipMemcpyHostToDevice);
	hipMemcpy(gpuMatrix, &_camera->_matrix, sizeof(mat4x4), hipMemcpyHostToDevice);
	hipMemcpy(gpuProjection, &_camera->projection, sizeof(mat4x4), hipMemcpyHostToDevice);
	hipMemcpy(gpuW, &((SDLGameEngine*)_this)->screen.SCREEN_WIDTH, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpuH, &((SDLGameEngine*)_this)->screen.SCREEN_HEIGHT, sizeof(int), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(gpuVerts, &_mesh->vertices[0], size * sizeof(vec3d), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		((SDLGameEngine*)_this)->throwException("cude.cu", __FUNCTION__, "memcpy: " + string(hipGetErrorString(cudaStatus)), false);
		goto Error;
	}

	calculatecuda<<<1, size>>>(gpuVerts, gpuOutVerts, gpuTemp, gpuWorld, gpuMatrix, gpuProjection, gpuW, gpuH);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		((SDLGameEngine*)_this)->throwException("cude.cu", __FUNCTION__, "gpu calculation launch failed: " + string(hipGetErrorString(cudaStatus)), false);
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		((SDLGameEngine*)_this)->throwException("cude.cu", __FUNCTION__, "hipDeviceSynchronize returned error code " + to_string(cudaStatus) + " after launching gpu calculation !", false);
		goto Error;
	}

	vec3d* src = gpuOutVerts;
	for (Vertvec::iterator it = _mesh->vertices.begin(); it != _mesh->vertices.end(); ++it) {
		vec3d dst;

		hipMemcpy(&dst, &src, sizeof(vec3d), hipMemcpyDeviceToHost);

		_verts->push_back(dst);

		src += sizeof(vec3d);
	}

	goto Error;

Error:
	hipFree(gpuWorld);
	hipFree(gpuMatrix);
	hipFree(gpuProjection);
	hipFree(gpuVerts);
	hipFree(gpuOutVerts);
	hipFree(gpuTemp);
	hipFree(gpuW);
	hipFree(gpuH);
}