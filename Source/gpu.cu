#include "hip/hip_runtime.h"
#include "game.h"

__global__ void SDLGameEngine::calculateGPU(vec3d* vert, mat4x4* World, mat4x4* matrix, mat4x4* projection) {
	int i = threadIdx.x;

	vec3d Transformed = MultiplyMatrixVector(*World, vert[i]);

	vec3d Viewed = MultiplyMatrixVector(*matrix, Transformed);

	vert[i] = MultiplyMatrixVector(*projection, Viewed);

	vert[i] = Vector_Div(vert[i], vert[i].w);
	vert[i].x += 1.0f;
	vert[i].y += 1.0f;
	vert[i].x *= 0.5f * (float)screen.SCREEN_WIDTH;
	vert[i].y *= 0.5f * (float)screen.SCREEN_HEIGHT;

}
void SDLGameEngine::_calculatePolygons(__parameters _param) {

	vector<vec3d>* _verts = _param.verts;
	mesh* _mesh = _param._mesh;
	camera* _camera = _param._camera;
	light* _light = _param._light;

	mat4x4 World, Rx, Ry, Rz, t;

	_param.Rx = &Rx;
	_param.Ry = &Ry;
	_param.Rz = &Rz;
	_param.t = &t;
	_param.World = &World;

	_meshApplyRotations(_param);
	_meshApplyTransations(_param);

	World = _matrixMakeIdentity();
	World = Matrix_MultiplyMatrix(Ry, Rz);
	World = Matrix_MultiplyMatrix(World, Rx);
	World = Matrix_MultiplyMatrix(World, t);

	mat4x4 _cameraRotationX;
	mat4x4 _cameraRotationY;
	mat4x4 _cameraRotationZ;
	mat4x4 _cameraRotation;

	_param.Rx = &_cameraRotationX;
	_param.Ry = &_cameraRotationY;
	_param.Rz = &_cameraRotationZ;

	_cameraApplyRotations(_param);

	_cameraRotation = Matrix_MultiplyMatrix(_cameraRotationZ, _cameraRotationY);
	_cameraRotation = Matrix_MultiplyMatrix(_cameraRotation, _cameraRotationZ);

	_camera->_up = { 0, 1, 0 };
	_camera->_target = { 0, 0, 1 };
	_camera->_pointAt = MultiplyMatrixVector(_cameraRotation, _camera->_target);

	_camera->_target = Vector_Add(_camera->position, _camera->_pointAt);

	mat4x4 __matrix = Matrix_PointAt(_camera->position, _camera->_target, _camera->_up);
	_camera->_matrix = __matrix; Matrix_QuickInverse(__matrix);

	mat4x4* gpuWorld;
	mat4x4* gpuMatrix;
	mat4x4* gpuProjection;
	vec3d* gpuVerts;

	size_t size = _mesh->vertices.size();

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	cudaStatus = hipMalloc((void**)&gpuWorld, sizeof(mat4x4));
	cudaStatus = hipMalloc((void**)&gpuMatrix, sizeof(mat4x4));
	cudaStatus = hipMalloc((void**)&gpuProjection, sizeof(mat4x4));
	cudaStatus = hipMalloc((void**)&gpuVerts, size * sizeof(vec3d));
	cudaStatus = hipMemcpy(gpuWorld, &World, sizeof(mat4x4), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(gpuMatrix, &_camera->_matrix, sizeof(mat4x4), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(gpuProjection, &_camera->projection, sizeof(mat4x4), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(gpuVerts, _mesh->vertices.begin()._Ptr, size * sizeof(vec3d), hipMemcpyHostToDevice);

	calculateGPU << < 1, size >> > (gpuVerts, gpuWorld, gpuMatrix, gpuProjection);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "gpu calculation launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching gpu calculation!\n", cudaStatus);
		goto Error;
	}

	for (int i = 0; i < size; i++) {
		vec3d v;
		cudaStatus = hipMemcpy(&v, (i * sizeof(vec3d)) + gpuVerts, sizeof(vec3d), hipMemcpyDeviceToHost);
		_verts->push_back(v);
	}
Error:
	hipFree(gpuWorld);
	hipFree(gpuMatrix);
	hipFree(gpuProjection);
	hipFree(gpuVerts);
}